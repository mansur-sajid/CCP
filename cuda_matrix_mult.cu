#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 10000
#define BLOCK_SIZE 16

__global__ void matrixMultiply(int *a, int *b, int *c, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        int sum = 0;
        for (int i = 0; i < width; ++i) {
            sum += a[row * width + i] * b[i * width + col];
        }
        c[row * width + col] = sum;
    }
}

int main() {
    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;
    size_t size = N * N * sizeof(int);

    // Allocate host memory
    h_a = (int *)malloc(size);
    h_b = (int *)malloc(size);
    h_c = (int *)malloc(size);

    // Initialize matrices
    for (int i = 0; i < N * N; ++i) {
        h_a[i] = 1;
        h_b[i] = 2;
    }

    // Allocate device memory
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy matrices from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Set up grid and block sizes
    dim3 dimGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);

    // Launch kernel
    matrixMultiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);

    // Record the stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Copy result from device to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Calculate elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Time taken: %f seconds\n", elapsedTime/1000);
    printf("Last element of resultant matrix: %d\n", h_c[N * N - 1]);

    // Free CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
